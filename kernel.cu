#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>

#define TX 32
#define TY 32
#define DEG_TO_RAD 0.0174532925

__device__
unsigned char clip(int n) { return n > 255 ? 255 : (n < 0 ? 0 : n); }

//  __device__
// float saturate(float t) { return t > 1.0f ? 1.0f : t < 0.0f ? 0.0f : t;}

 __global__
 void imagePass(uchar4 *d_out, int w, int h, Settings settings) {
    const int c = blockIdx.x * blockDim.x + threadIdx.x;
    const int r = blockIdx.y * blockDim.y + threadIdx.y;
    
	if ((c >= w) || (r >= h)) return; // Check if within image bounds
    
	const int i = c + r * w; // 1D indexing

    int index = 0;
    int OIX, OIY;
    int sum = 0;
    for(int offsetX = -1; offsetX <= 1; offsetX++){
        for(int offsetY = -1; offsetY <= 1; offsetY++){
            OIX = (c + offsetX);// + (r + offsetY) * w;
            OIY = (r + offsetY);
            if ((OIX >= w) || (OIY >= h) || (OIX < 0) || (OIY < 0)) break; 

            index = OIX + OIY * w;
            sum += d_out[index].x;
        }
    }

    float blurredCol = (sum / 9) / 255.0f;
    float color = d_out[i].x / 255.0f;

    float diffuseWeight = __saturatef(settings.diffuseRate * settings.deltaTime);
    blurredCol = color * (1 - diffuseWeight) + blurredCol * diffuseWeight;

    float decayRate = settings.decayRate;
    color = max(0.0, blurredCol - decayRate * settings.deltaTime);


    //d_out[i].x = sum;
    //d_out[i].y = sum;//clip(d_out[i].y - 1);
    //d_out[i].z = sum;//clip(d_out[i].z - 1);
    
	d_out[i].x = clip(color * 255);
    d_out[i].y = clip(color * 255);
    d_out[i].z = clip(color * 255);
    d_out[i].w = 255;
 }

__device__
float sense(Agent agent, float sensorAngelOffset, uchar4* d_out, int w, int h, Settings settings){
    float sensorAngle = agent.angle + sensorAngelOffset;
    float sensorDirx = cosf(sensorAngle);
    float sensorDiry = sinf(sensorAngle);

    int sensorCenterx = (agent.x * w) + sensorDirx * settings.sensorOffsetDistance;
    int sensorCentery = (agent.y * h) + sensorDiry * settings.sensorOffsetDistance;

    int sum = 0;
    int sensorSize = 1;//settings.sensorSize;

    for (int offsetX = -sensorSize; offsetX <= sensorSize; offsetX++){
        for (int offsetY = -sensorSize; offsetY <= sensorSize; offsetY++){
            int posx = min(w - 1, max(0, sensorCenterx + offsetX));//sensorCenterx + offsetX;
            int posy = min(h - 1, max(0, sensorCentery + offsetY));

            if (posx >= 0 && posx < w && posy > 0 && posy < h){
                int idx = posx + posy * w;
                sum += d_out[idx].x;
            }
        }
    }

    return sum;
}

__global__
void agentKernel(hiprandState* state, Agent* agents, uchar4* d_out, int numAgents, int w, int h, Settings settings) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i > numAgents) return;

    hiprandState localState = state[i];

    Agent agent = agents[i];

    float sensorAngleSpacing = settings.sensorAngleSpacing * DEG_TO_RAD;

    float weightForward = sense(agent, 0, d_out, w, h, settings);
    float weightLeft = sense(agent, sensorAngleSpacing, d_out, w, h, settings);
    float weightRight = sense(agent, -sensorAngleSpacing, d_out, w, h, settings);

    float randomSteerStrength = hiprand_uniform(&localState);
    float turnSpeed = settings.turnSpeed * DEG_TO_RAD;

    if ((weightForward > weightLeft) && (weightForward > weightRight)){
        agent.angle += 0;
    }
    else if ((weightForward < weightLeft) && (weightForward < weightRight)){
        agent.angle = agent.angle + (randomSteerStrength - 0.5) * 2 * turnSpeed;// * 0.1f;
    }
    else if (weightRight > weightLeft){
        agent.angle = agent.angle - randomSteerStrength * turnSpeed;// * 0.1f;
    }
    else if (weightLeft > weightRight){
        agent.angle = agent.angle + randomSteerStrength * turnSpeed;// * 0.1f;
    }

    float directionx = cosf(agent.angle);
    float directiony = sinf(agent.angle);
    float newPosx = agent.x + directionx * settings.deltaTime * settings.agentSpeed;//0.00025f;
    float newPosy = agent.y + directiony * settings.deltaTime * settings.agentSpeed;

    if (newPosx < 0 || newPosx >= 1 || newPosy < 0 || newPosy >= 1){
        newPosx = min(1.0 - 0.001, max(0.0, newPosx));
        newPosy = min(1.0 - 0.001, max(0.0, newPosy));
        agent.angle = hiprand_uniform(&localState) * 2.0f * 3.14159f;
    }

    agent.x = newPosx;
    agent.y = newPosy;

    agents[i] = agent;
    if (agent.x > 1.0 || agent.y > 1.0 || agent.x < 0.0 || agent.y < 0.0) return;

    int agentx = agent.x * w;
    int agenty = agent.y * h;

    int id = agentx + agenty * w;
    if (id < 0 || id > (w * h)) return;
    d_out[id].x = 255;
    d_out[id].y = 255;
    d_out[id].z = 255;
 }

 __global__
 void setupKernel(hiprandState* devStates, Agent* agents, int numAgents, int w, int h){
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i > numAgents) return;
    
    hiprand_init(1548, i, 0, &devStates[i]);

    hiprandState localState = devStates[i];
    float aspect = (float)w / h;

    float r = 0.25f * sqrtf(hiprand_uniform(&localState)); // float from 0.0 to 1.0f
    float theta = hiprand_uniform(&localState) * 2.0f * 3.1415f;

    float x = 0.5f + r * cosf(theta);
    float y = 0.5f + r * sinf(theta);

    float dx = 0.5 - x;
    float dy = 0.5 - y;
    float angle = atan2f(dy, dx);

    agents[i].x = x;
    agents[i].y = y * aspect - 0.40;
    agents[i].angle = angle;//hiprand_uniform(&localState);
 }

 void kernelLauncher(uchar4 *d_out, int w, int h, Settings settings) {
    const dim3 gridSize = dim3((w + TX - 1)/TX, (h + TY - 1)/TY);
    const dim3 blockSize(TX, TY);
    imagePass<<<gridSize, blockSize>>>(d_out, w, h, settings);
 }
// TS = 0.1 SS = 0.2
 void kernelLauncherAgent(hiprandState* states, Agent* agents, int numAgents, uchar4* d_out, int w, int h, Settings settings){
    const dim3 gridSize(2048, 1, 1);
    const dim3 blockSize(512, 1, 1);
    agentKernel<<<gridSize, blockSize>>>(states, agents, d_out, numAgents, w, h, settings);
 }

 void KernelLauncherSetup(hiprandState* state, Agent* agents, int numAgents, int w, int h){
    const dim3 gridSize(2048, 1, 1);
    const dim3 blockSize(512, 1, 1);
    setupKernel<<<gridSize, blockSize>>>(state, agents, numAgents, w, h);
 }
